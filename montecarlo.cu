#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include "helper_image.h"
#include "helper_string.h"
#include "helper_timer.h"
#include "exception.h"

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	2048
#endif

// setting the number of trials in the monte carlo simulation:
#ifndef BLOCKSIZE
#define BLOCKSIZE	32
#endif


// ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	40.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	30.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	50.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 70.0;	// cannonball launch angle in degrees
const float THMAX =	80.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL = 5.0;		// tolerance in cannonball hitting the castle in meters
				// castle is destroyed if cannonball lands between d-TOL and d+TOL


// function prototypes
void        CudaCheckError( );
float		Ranf( float, float );
void        TimeOfDaySeed( );

// degrees-to-radians -- callable from the device:
__device__
float
Radians( float d )
{
        return (M_PI/180.f) * d;
}

// the kernel:
__global__
void
MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
        unsigned int gid      = blockIdx.x*blockDim.x + threadIdx.x;

        // randomize everything:
        float v   = dvs[gid];
        float thr = Radians( dths[gid] );
        float vx  = v * cos(thr);
        float vy  = v * sin(thr);
        float  g  =  dgs[gid];
        float  h  =  dhs[gid];
        float  d  =  dds[gid];

        int numHits = 0;

        // see if the ball doesn't even reach the cliff:
        float t = -vy / ( 0.5*GRAVITY );
        float x = vx * t;
        if( x > g )
        {
	        t = g / vx;
			float y = (vy * t) + (0.5 * GRAVITY * (t * t));

            // see if ball hits the vertical cliff face
            if ( y > h )
            {
                // the ball hits the upper deck:
                // the time solution for this is a quadratic equation of the form:
                // at^2 + bt + c = 0.
                // where 'a' multiplies time^2
                //       'b' multiplies time
                //       'c' is a constant
                float a = 0.5 * GRAVITY;
                float b = vy;
                float c = -h;
                float disc = b*b - 4.f*a*c;	// quadratic formula discriminant

                // ball doesn't go as high as the upper deck:
				// this should "never happen" ... :-)
                if ( disc >= 0. )
                {
                    // successfully hits the ground above the cliff:
                    // get the intersection:
                    disc = sqrtf( disc );
                    float t1 = (-b + disc ) / ( 2.f*a );	// time to intersect high ground
                    float t2 = (-b - disc ) / ( 2.f*a );	// time to intersect high ground

                    // only care about the second intersection
                    float tmax = t1;
                    if ( t2 > t1 )
                        tmax = t2;

                    // how far does the ball land horizontlly from the edge of the cliff?
                    float upperDist = vx * tmax  -  g;

                    // see if the ball hits the castle:
                    if(  fabs( upperDist - d ) <= TOL )
                    {
                        fprintf(stdout, "Hit!\n");
                        numHits = 1;
                    }
                } 

            }
        }

        dhits[gid] = numHits;
}


// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT

int
main( int argc, char* argv[ ] )
{
        TimeOfDaySeed( );

        int NUMBLOCKS = NUMTRIALS / BLOCKSIZE;

        int dev = findCudaDevice(argc, (const char **)argv);

        // better to define these here so that the rand() calls don't get into the thread timing:
        float *hvs   = new float [NUMTRIALS];
        float *hths  = new float [NUMTRIALS];
        float *hgs   = new float [NUMTRIALS];
        float *hhs   = new float [NUMTRIALS];
        float *hds   = new float [NUMTRIALS];
        int   *hhits = new int   [NUMTRIALS];

        // fill the random-value arrays:
        for( int n = 0; n < NUMTRIALS; n++ )
        {
            hvs[n]  = Ranf(  VMIN,  VMAX );
            hths[n] = Ranf( THMIN, THMAX );
            hgs[n]  = Ranf(  GMIN,  GMAX );
            hhs[n]  = Ranf(  HMIN,  HMAX );
            hds[n]  = Ranf(  DMIN,  DMAX );
        }



        // allocate device memory:
        float *dvs, *dths, *dgs, *dhs, *dds;
        int   *dhits;

        hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
        hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
        CudaCheckError( );

        // copy host memory to the device:
        hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        CudaCheckError( );

        // setup the execution parameters:
        dim3 grid( NUMBLOCKS, 1, 1 );
        dim3 threads( BLOCKSIZE, 1, 1 );

        // allocate cuda events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop  );
        CudaCheckError( );

        // let the gpu go quiet:
        hipDeviceSynchronize( );

        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

        // execute the kernel:
        MonteCarlo<<< grid, threads >>>( IN dvs, IN dths, IN dgs, IN dhs, IN dds,   OUT dhits );

        // record the stop event:
        hipEventRecord( stop, NULL );
        CudaCheckError( );

        // wait for the stop event to complete:
        hipDeviceSynchronize( );
        hipEventSynchronize( stop );
        CudaCheckError( );

        float msecTotal = 0.0f;
        hipEventElapsedTime( &msecTotal, start, stop );
        CudaCheckError( );

        // compute and print the performance



        // copy result from the device to the host:
        hipMemcpy( hhits, dhits, NUMTRIALS*sizeof(int), hipMemcpyDeviceToHost );
        CudaCheckError( );

        // add up the hhits[ ] array: :
        int totalHits = 0;
        for (int i = 0; i < NUMTRIALS; i++ )
        {
            totalHits += hhits[i];
        }

        // compute and print the probability:

        double probability = totalHits / NUMTRIALS;
        fprintf(stdout, "Probability = %6.2lf\n", 100*probability);

        // clean up host memory:
        delete [ ] hvs;
        delete [ ] hths;
        delete [ ] hgs;
        delete [ ] hhs;
        delete [ ] hds;
        delete [ ] hhits;

        // clean up device memory:
        hipFree( dvs );
        hipFree( dths );
        hipFree( dgs );
        hipFree( dhs );
        hipFree( dds );
        hipFree( dhits );
        CudaCheckError( );

	return 0;
}

void
CudaCheckError( )
{
        hipError_t e = hipGetLastError( );
        if( e != hipSuccess )
        {
                fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
        }
}

float
Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

// call this if you want to force your program to use
// a different random number sequence every time you run it:
void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}